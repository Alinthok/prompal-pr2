#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void matrixMultiplySharedKernel(float *A, float *B, float *C, int N) {
    __shared__ float s_A[32][32];
    __shared__ float s_B[32][32];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    for (int k = 0; k < (N + 32 - 1) / 32; k++) {
        if (k * 32 + threadIdx.x < N && row < N) {
            s_A[threadIdx.y][threadIdx.x] = A[row * N + k * 32 + threadIdx.x];
        } else {
            s_A[threadIdx.y][threadIdx.x] = 0.0;
        }
        if (k * 32 + threadIdx.y < N && col < N) {
            s_B[threadIdx.y][threadIdx.x] = B[(k * 32 + threadIdx.y) * N + col];
        } else {
            s_B[threadIdx.y][threadIdx.x] = 0.0;
        }

        __syncthreads();

        for (int n = 0; n < 32; ++n) {
            sum += s_A[threadIdx.y][n] * s_B[n][threadIdx.x];
        }

        __syncthreads();
    }

    if (row < N && col < N) {
        C[row * N + col] = sum;
    }
}

double parallelMatrixMultiplyShared(float *A, float *B, float *C, int N, int blockSize) {
    float *d_A, *d_B, *d_C;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    dim3 dimBlock(blockSize, blockSize);
    dim3 dimGrid((N + blockSize - 1) / blockSize, (N + blockSize - 1) / blockSize);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventRecord(start, 0);

    matrixMultiplySharedKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, N);

    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Elapsed time GPU: %f ms\n", elapsedTime);

    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return elapsedTime;
}

int main() {
    int N = 1024; // Change as needed
    int blockSize = 32; // Change as needed

    printf("Masukan N dan blockSize\n");
    scanf("%d %d", &N, &blockSize);
    printf("N: %d, Block size: %d\n", N, blockSize);

    float *A = (float*)malloc(N * N * sizeof(float));
    float *B = (float*)malloc(N * N * sizeof(float));
    float *C = (float*)malloc(N * N * sizeof(float));

    // Initialize matrices A and B
    for (int i = 0; i < N * N; i++) {
        A[i] = rand() % 100;
        B[i] = rand() % 100;
    }

    clock_t begin = clock();

    double gpuTime;

    gpuTime = parallelMatrixMultiplyShared(A, B, C, N, blockSize);

    clock_t end = clock();
    double cpuTime = (double)(end-begin)/(CLOCKS_PER_SEC/1000);

    printf("Elapsed time on CPU: %f ms\n", cpuTime);

    printf("COMM TIME = CPU-GPU = %f ms\n", cpuTime-gpuTime);

    printf("x/y = %f\n", gpuTime/(cpuTime-gpuTime));

    printf("SMALL PART OF MATRIX:\n");

    // Print a small part of the matrix to verify correctness
    for (int i = 0; i < 5; i++) {
        for (int j = 0; j < 5; j++) {
            printf("%f ", C[i * N + j]);
        }
        printf("\n");
    }

    free(A);
    free(B);
    free(C);

    return 0;
}